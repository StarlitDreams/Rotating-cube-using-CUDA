#include "hip/hip_runtime.h"
#include "screen.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <iostream>

/**
 * CUDA error checking macro
 * Wraps CUDA API calls and checks for errors
 */
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**
 * Helper function for CUDA error checking
 * Reports errors to stderr and optionally aborts execution
 */
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) { 
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * Plain POD structure for device/host transfer
 * Avoids including SDL types in device code
 */
struct PointF { 
    float x, y; 
};

/**
 * 3D vector structure for representing points in 3D space
 */
struct vec3 {
    float x, y, z;
};

/**
 * Edge connection structure defining relationships between vertices
 */
struct connection {
    int a, b;
};

/**
 * CUDA kernel for 3D rotation transformations
 * Applies sequential rotations around X, Y, and Z axes
 * 
 * @param outVec Output array of rotated vectors
 * @param inVec Input array of vectors to rotate
 * @param n Number of vectors
 * @param x Rotation angle around X axis (radians)
 * @param y Rotation angle around Y axis (radians)
 * @param z Rotation angle around Z axis (radians)
 */
__global__ void rotate(vec3* outVec, const vec3* inVec, int n, float x, float y, float z) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    float px = inVec[idx].x;
    float py = inVec[idx].y;
    float pz = inVec[idx].z;

    float rad = x;
    float temp_y = cosf(rad) * py - sinf(rad) * pz;
    float temp_z = sinf(rad) * py + cosf(rad) * pz;
    py = temp_y;
    pz = temp_z;

    rad = y;
    float temp_x = cosf(rad) * px + sinf(rad) * pz;
    temp_z = -sinf(rad) * px + cosf(rad) * pz;
    px = temp_x;
    pz = temp_z;

    rad = z;
    temp_x = cosf(rad) * px - sinf(rad) * py;
    temp_y = sinf(rad) * px + cosf(rad) * py;
    px = temp_x;
    py = temp_y;

    outVec[idx].x = px;
    outVec[idx].y = py;
    outVec[idx].z = pz;
}

/**
 * CUDA kernel for parallel line rasterization
 * Uses DDA-style parametric interpolation with one thread per pixel
 * 
 * @param outPts Output array of rasterized points
 * @param n Number of points to generate
 * @param x0 Starting X coordinate
 * @param y0 Starting Y coordinate
 * @param x1 Ending X coordinate
 * @param y1 Ending Y coordinate
 */
__global__ void rasterizeLine(PointF* outPts, int n, float x0, float y0, float x1, float y1) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float t = (n > 1) ? (float)i / (float)(n - 1) : 0.0f;
    outPts[i].x = x0 + t * (x1 - x0);
    outPts[i].y = y0 + t * (y1 - y0);
}

/**
 * Main application entry point
 * Renders a rotating 3D cube using CUDA-accelerated transformations and line rasterization
 */
int main() {
    Screen screen;
    
    std::vector<vec3> points = {
        {810, 390, -150},
        {1110, 390, -150},
        {1110, 690, -150},
        {810, 690, -150},
        {810, 390, 150},
        {1110, 390, 150},
        {1110, 690, 150},
        {810, 690, 150}
    };

    std::vector<connection> connections = {
        {0, 1}, {1, 2}, {2, 3}, {3, 0},
        {4, 5}, {5, 6}, {6, 7}, {7, 4},
        {0, 4}, {1, 5}, {2, 6}, {3, 7}
    };
    
    vec3 c{0, 0, 0};
    for (auto p : points) {
        c.x += p.x;
        c.y += p.y;
        c.z += p.z;
    }
    c.x /= points.size();
    c.y /= points.size();
    c.z /= points.size();

    std::vector<vec3> centeredPoints(points.size());
    for (size_t i = 0; i < points.size(); i++) {
        centeredPoints[i].x = points[i].x - c.x;
        centeredPoints[i].y = points[i].y - c.y;
        centeredPoints[i].z = points[i].z - c.z;
    }

    vec3* d_inPoints = nullptr, * d_outPoints = nullptr;
    const int numPoints = static_cast<int>(points.size());
    cudaCheck(hipMalloc(&d_inPoints, numPoints * sizeof(vec3)));
    cudaCheck(hipMalloc(&d_outPoints, numPoints * sizeof(vec3)));

    cudaCheck(hipMemcpy(d_inPoints, centeredPoints.data(), numPoints * sizeof(vec3), hipMemcpyHostToDevice));

    const int TPB = 128;
    int blocks = (numPoints + TPB - 1) / TPB;

    float rotX = 0.0f, rotY = 0.0f, rotZ = 0.0f;

    while (screen.input()) {
        screen.clear_points();

        rotX += 0.01f;
        rotY += 0.02f;
        rotZ += 0.005f;

        rotate<<<blocks, TPB>>>(d_outPoints, d_inPoints, numPoints, rotX, rotY, rotZ);
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());

        std::vector<vec3> rotatedPoints(numPoints);
        cudaCheck(hipMemcpy(rotatedPoints.data(), d_outPoints, numPoints * sizeof(vec3), hipMemcpyDeviceToHost));

        std::vector<SDL_FPoint> displayPoints;
        displayPoints.reserve(numPoints);
        for (int i = 0; i < numPoints; i++) {
            displayPoints.push_back(SDL_FPoint{
                rotatedPoints[i].x + c.x,
                rotatedPoints[i].y + c.y
            });
        }

        screen.pixels(displayPoints.data(), displayPoints.size());

        for (const auto& conn : connections) {
            float x0 = displayPoints[conn.a].x;
            float y0 = displayPoints[conn.a].y;
            float x1 = displayPoints[conn.b].x;
            float y1 = displayPoints[conn.b].y;
            
            float dx = fabsf(x1 - x0), dy = fabsf(y1 - y0);
            int n = (int)fmaxf(dx, dy) + 1;
            if (n < 2) n = 2;

            PointF* d_pts = nullptr;
            cudaCheck(hipMalloc(&d_pts, n * sizeof(PointF)));
            
            int lineBlocks = (n + TPB - 1) / TPB;
            
            rasterizeLine<<<lineBlocks, TPB>>>(d_pts, n, x0, y0, x1, y1);
            cudaCheck(hipGetLastError());
            cudaCheck(hipDeviceSynchronize());

            std::vector<PointF> h_pts(n);
            cudaCheck(hipMemcpy(h_pts.data(), d_pts, n * sizeof(PointF), hipMemcpyDeviceToHost));
            cudaCheck(hipFree(d_pts));

            std::vector<SDL_FPoint> sdlPts;
            sdlPts.reserve(h_pts.size());
            for (const auto& p : h_pts) {
                sdlPts.push_back(SDL_FPoint{p.x, p.y});
            }
            screen.pixels(sdlPts.data(), sdlPts.size());
        }

        screen.show();
        SDL_Delay(7);
    }

    cudaCheck(hipFree(d_inPoints));
    cudaCheck(hipFree(d_outPoints));

    return 0;
}